/***************************************************************************************************
 * Copyright (c) 2024 - 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
#include <cstdlib>
#include <cstdio>
#include <cassert>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cute/tensor.hpp>

#include "cutlass/cluster_launch.hpp"
#include "cutlass/util/print_error.hpp"
#include "cutlass/util/GPU_Clock.hpp"
#include "cutlass/util/helper_cuda.hpp"
#include "cutlass/arch/mma_sm90.h"
#include "cutlass/device_kernel.h"

#include "fprint_mat.h"
#include "gemm.h"

// Setup params for an NT GEMM
template <class TA, class TB, class TC,
          class Alpha, class Beta>
void
gemm_nt(int m, int n, int k,
        Alpha alpha,
        TA const* A, int ldA,
        TB const* B, int ldB,
        Beta beta,
        TC      * C, int ldC,
        hipStream_t stream = 0)
{
  // Define shapes (dynamic)
  auto M = int(m);
  auto N = int(n);
  auto K = int(k);
  auto prob_shape = make_shape(M, N, K);                     // (M, N, K)

  // Define TN strides (mixed)
  auto dA = make_stride(Int<1>{}, ldA);                      // (dM, dK)
  auto dB = make_stride(Int<1>{}, ldB);                      // (dN, dK)
  auto dC = make_stride(Int<1>{}, ldC);                      // (dM, dN)

  // Define CTA tile sizes (static)
  auto bM = Int<128>{};
  auto bN = Int<128>{};
  auto bK = Int< 64>{};
  auto cta_tiler = make_shape(bM, bN, bK);                   // (BLK_M, BLK_N, BLK_K)
  auto bP = Int<  3>{};  // Pipeline

  // Define the smem layouts (static)
  /*
  GMMA::Layout_MN_SW128_Atom<TA> expands to
  using cute::SM90::GMMA::Layout_MN_SW128_Atom<TA> = 
    cute::ComposedLayout<
      cute::Swizzle<3, 4, 3>, 
      cute::smem_ptr_flag_bits<16>, 
      cute::Layout<cute::Shape<cute::_64, cute::_8>, cute::Stride<cute::_1, cute::_64>>
    > 
  */
  auto sA = tile_to_shape(GMMA::Layout_MN_SW128_Atom<TA>{}, make_shape(bM,bK,bP));
  printf("sA: "); cute::print(sA); printf("\n");

  auto sB = tile_to_shape(GMMA::Layout_MN_SW128_Atom<TB>{}, make_shape(bN,bK,bP));

  // Define the MMA
  TiledMMA tiled_mma = make_tiled_mma(SM90_64x64x16_F16F16F16_SS<GMMA::Major::MN,GMMA::Major::MN>{});

  // Define the TMAs
  // Create Global memory tensors for TMA inspection
  Tensor mA = make_tensor(A, make_shape(M,K), dA);
  Tensor mB = make_tensor(B, make_shape(N,K), dB);

  // Create TMA Atoms with the desired copy operation on the source and destination
  Copy_Atom tmaA = make_tma_atom(SM90_TMA_LOAD{}, mA, sA(_,_,0), make_shape(bM,bK));
  Copy_Atom tmaB = make_tma_atom(SM90_TMA_LOAD{}, mB, sB(_,_,0), make_shape(bN,bK));

  //
  // Setup and Launch
  //

  // Launch parameter setup
  int smem_size = int(sizeof(sm90::SharedStorage<TA, TB, decltype(sA), decltype(sB)>));
  printf("smem_size: %d\n", smem_size);

  dim3 dimBlock(size(tiled_mma)); // 128
  dim3 dimCluster(2, 1, 1);
  dim3 dimGrid(round_up(size(ceil_div(m, bM)), dimCluster.x),
               round_up(size(ceil_div(n, bN)), dimCluster.y));
  cutlass::ClusterLaunchParams params = {dimGrid, dimBlock, dimCluster, smem_size};

  void const* kernel_ptr = reinterpret_cast<void const*>(
                              &sm90::gemm_device<decltype(prob_shape), decltype(cta_tiler),
                                           TA, decltype(sA), decltype(tmaA),
                                           TB, decltype(sB), decltype(tmaB),
                                           TC, decltype(dC), decltype(tiled_mma),
                                           decltype(alpha), decltype(beta)>);

  CUTE_CHECK_ERROR(hipFuncSetAttribute(reinterpret_cast<const void*>(
    kernel_ptr),
    hipFuncAttributeMaxDynamicSharedMemorySize,
    smem_size));

  // Kernel Launch
  cutlass::Status status = cutlass::launch_kernel_on_cluster(params, kernel_ptr,
                                                             prob_shape, cta_tiler,
                                                             A, tmaA,
                                                             B, tmaB,
                                                             C, dC, tiled_mma,
                                                             alpha, beta);
  CUTE_CHECK_LAST();

  if (status != cutlass::Status::kSuccess) {
    std::cerr << "Error: Failed at kernel Launch" << std::endl;
  }
}

// Setup params for a TN GEMM
template <class TA, class TB, class TC,
          class Alpha, class Beta>
void
gemm_tn(int m, int n, int k,
        Alpha alpha,
        TA const* A, int ldA,
        TB const* B, int ldB,
        Beta beta,
        TC      * C, int ldC,
        hipStream_t stream = 0)
{
  // Define shapes (dynamic)
  auto M = int(m);
  auto N = int(n);
  auto K = int(k);
  auto prob_shape = make_shape(M, N, K);                     // (M, N, K)

  // Define TN strides (mixed)
  auto dA = make_stride(ldA, Int<1>{});                      // (dM, dK)
  auto dB = make_stride(ldB, Int<1>{});                      // (dN, dK)
  auto dC = make_stride(Int<1>{}, ldC);                      // (dM, dN)

  // Define CTA tile sizes (static)
  auto bM = Int<128>{};
  auto bN = Int<128>{};
  auto bK = Int< 64>{};
  auto cta_tiler = make_shape(bM, bN, bK);                   // (BLK_M, BLK_N, BLK_K)
  auto bP = Int<3>{};  // Pipeline

  // Define the smem layouts (static)
  auto sA = tile_to_shape(GMMA::Layout_K_SW128_Atom<TA>{}, make_shape(bM,bK,bP));
  auto sB = tile_to_shape(GMMA::Layout_K_SW128_Atom<TB>{}, make_shape(bN,bK,bP));

  // Define the MMA
  TiledMMA tiled_mma = make_tiled_mma(SM90_64x64x16_F16F16F16_SS<GMMA::Major::K,GMMA::Major::K>{});
  /*
  struct cute::TiledMMA<
    cute::MMA_Atom<cute::SM90_64x64x16_F16F16F16_SS<cute::SM90::GMMA::Major::K, cute::SM90::GMMA::Major::K>>, 
    cute::Layout<cute::Shape<cute::_1, cute::_1, cute::_1>>, 
    cute::Tile<cute::Underscore, cute::Underscore, cute::Underscore>
  >
  */

  // Define the TMAs
  // Create Global memory tensors for TMA inspection
  Tensor mA = make_tensor(A, make_shape(M,K), dA);
  Tensor mB = make_tensor(B, make_shape(N,K), dB);

  // Create TMA Atoms with the desired copy operation on the source and destination
  Copy_Atom tmaA = make_tma_atom(SM90_TMA_LOAD{}, mA, sA(_,_,0), make_shape(bM,bK));
  Copy_Atom tmaB = make_tma_atom(SM90_TMA_LOAD{}, mB, sB(_,_,0), make_shape(bN,bK));

  //
  // Setup and Launch
  //

  // Launch parameter setup
  int smem_size = int(sizeof(sm90::SharedStorage<TA, TB, decltype(sA), decltype(sB)>));
  dim3 dimBlock(size(tiled_mma));
  dim3 dimCluster(2, 1, 1);
  dim3 dimGrid(round_up(size(ceil_div(m, bM)), dimCluster.x),
               round_up(size(ceil_div(n, bN)), dimCluster.y));
  cutlass::ClusterLaunchParams params = {dimGrid, dimBlock, dimCluster, smem_size};

  void const* kernel_ptr = reinterpret_cast<void const*>(
                              &sm90::gemm_device<decltype(prob_shape), decltype(cta_tiler),
                                           TA, decltype(sA), decltype(tmaA),
                                           TB, decltype(sB), decltype(tmaB),
                                           TC, decltype(dC), decltype(tiled_mma),
                                           decltype(alpha), decltype(beta)>);

  CUTE_CHECK_ERROR(hipFuncSetAttribute(reinterpret_cast<const void*>(
    kernel_ptr),
    hipFuncAttributeMaxDynamicSharedMemorySize,
    smem_size));

  // Kernel Launch
  cutlass::Status status = cutlass::launch_kernel_on_cluster(params, kernel_ptr,
                                                             prob_shape, cta_tiler,
                                                             A, tmaA,
                                                             B, tmaB,
                                                             C, dC, tiled_mma,
                                                             alpha, beta);
  CUTE_CHECK_LAST();

  if (status != cutlass::Status::kSuccess) {
    std::cerr << "Error: Failed at kernel Launch" << std::endl;
  }
}

template <class TA, class TB, class TC,
          class Alpha, class Beta>
void
gemm(char transA, char transB, int m, int n, int k,
     Alpha alpha,
     TA const* A, int ldA,
     TB const* B, int ldB,
     Beta beta,
     TC      * C, int ldC,
     hipStream_t stream = 0)
{
  if (transA == 'N' && transB == 'T') {
    return gemm_nt(m, n, k, alpha, A, ldA, B, ldB, beta, C, ldC, stream);
  } else
  if (transA == 'T' && transB == 'N') {
    return gemm_tn(m, n, k, alpha, A, ldA, B, ldB, beta, C, ldC, stream);
  }
  assert(false && "Not implemented");
}

int main(int argc, char** argv)
{

  hipDeviceProp_t props;
  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (props.major != 9) {
    std::cout << "This example requires NVIDIA's Hopper Architecture GPU with compute capability 90a\n" << std::endl;
    return 0;
  }

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

  int m = 512;
  if (argc >= 2)
    sscanf(argv[1], "%d", &m);

  int n = 256;
  if (argc >= 3)
    sscanf(argv[2], "%d", &n);

  int k = 1024;
  if (argc >= 4)
    sscanf(argv[3], "%d", &k);

  char transA = 'N';
  if (argc >= 5)
    sscanf(argv[4], "%c", &transA);

  char transB = 'T';
  if (argc >= 6)
    sscanf(argv[5], "%c", &transB);

  using TA = cute::half_t;
  using TB = cute::half_t;
  using TC = cute::half_t;
  using TI = cute::half_t;

  TI alpha = TI(1.0f);
  TI beta  = TI(0.0f);

  thrust::host_vector<TA> h_A(m*k);
  thrust::host_vector<TB> h_B(n*k);
  thrust::host_vector<TC> h_C(m*n);

  // Initialize the tensors
  // for (int j = 0; j < m*k; ++j) h_A[j] = TA(int((rand() % 2) ? 1 : -1));
  // for (int j = 0; j < n*k; ++j) h_B[j] = TB(int((rand() % 2) ? 1 : -1));
  for (int j = 0; j < m*k; ++j) h_A[j] = TA(0.25);
  for (int j = 0; j < n*k; ++j) h_B[j] = TB(2.0);
  for (int j = 0; j < m*n; ++j) h_C[j] = TC(0);

  FILE* file_ptr = get_file_ptr("output.txt");
  fprint_mat(file_ptr, "h_A", h_A.data(), dim3(m, k, 1));
  fprintf(file_ptr, "\n\n");

  fprint_mat(file_ptr, "h_B", h_B.data(), dim3(n, k, 1));
  fprintf(file_ptr, "\n\n");

  thrust::device_vector<TA> d_A = h_A;
  thrust::device_vector<TB> d_B = h_B;
  thrust::device_vector<TC> d_C = h_C;

  int ldA = 0, ldB = 0, ldC = m;

  if (transA == 'N') {
    ldA = m;
  } else if (transA == 'T') {
    ldA = k;
  } else {
    assert(false);
  }

  if (transB == 'N') {
    ldB = k;
  } else if (transB == 'T') {
    ldB = n;
  } else {
    assert(false);
  }

  // Run once
  d_C = h_C;
  gemm(transA, transB, m, n, k,
       alpha,
       d_A.data().get(), ldA,
       d_B.data().get(), ldB,
       beta,
       d_C.data().get(), ldC);
  CUTE_CHECK_LAST();
  thrust::host_vector<TC> cute_result = d_C;

  fprint_mat(file_ptr, "cute_result", cute_result.data(), dim3(m, n, 1));

  // Timing iterations
  // double gflops = (2.0*m*n*k) * 1e-9;

  // const int timing_iterations = 100;
  // GPU_Clock timer;
  // timer.start();
  // for (int i = 0; i < timing_iterations; ++i) {
  //   gemm(transA, transB, m, n, k,
  //        alpha,
  //        d_A.data().get(), ldA,
  //        d_B.data().get(), ldB,
  //        beta,
  //        d_C.data().get(), ldC);
  // }
  // double cute_time = timer.seconds() / timing_iterations;
  // CUTE_CHECK_LAST();
  // printf("CUTE_GEMM:     [%6.1f]GFlop/s  (%6.4f)ms\n", gflops / cute_time, cute_time*1000);

#else

  std::cout << "CUTLASS_ARCH_MMA_SM90_SUPPORTED must be enabled, but it is not. Test is waived \n" << std::endl;
#endif

  printf("Output file: output.txt\n");
  return 0;

}
