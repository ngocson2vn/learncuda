
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdint>


#define CUDA_CHECK_ERROR(e)                                    \
do {                                                           \
  hipError_t code = (e);                                      \
  if (code != hipSuccess) {                                   \
    fprintf(stderr, "<%s:%d> %s:\n    %s: %s\n",               \
            __FILE__, __LINE__, #e,                            \
            hipGetErrorName(code), hipGetErrorString(code)); \
    fflush(stderr);                                            \
    exit(1);                                                   \
  }                                                            \
} while (0)

void device_init(int device_id, bool quiet = false) {
  hipDeviceProp_t device_prop;
  std::size_t    device_free_physmem;
  std::size_t    device_total_physmem;

  CUDA_CHECK_ERROR(hipSetDevice(device_id));
  CUDA_CHECK_ERROR(hipMemGetInfo(&device_free_physmem, &device_total_physmem));
  CUDA_CHECK_ERROR(hipGetDeviceProperties(&device_prop, device_id));

  if (device_prop.major < 1) {
    fprintf(stderr, "Device does not support CUDA.\n");
    exit(1);
  }

  if (!quiet) {
    printf("Using device %d: %s  (SM%d, %d SMs)\n",
           device_id, device_prop.name,
           device_prop.major * 10 + device_prop.minor,
           device_prop.multiProcessorCount);
    fflush(stdout);
  }
}


// Elect one thread in the warp. 
// The elected thread gets its predicate set to true, all others obtain false.
__device__ uint32_t elect_one_sync()
{
  uint32_t pred = 0;
  asm volatile(
    "{\n"
      ".reg .pred %%px;\n"
      "elect.sync _|%%px, %1;\n"
      "@%%px mov.s32 %0, 1;\n"
    "}\n"
    : "+r"(pred)
    : "r"(0xFFFFFFFF)
  );

  return pred;
}

__global__ void test_kernel() {
  uint32_t is_elected_lane = elect_one_sync();
  printf("threadIdx.x = %d is_elected_lane = %d\n", threadIdx.x, is_elected_lane);
}

int main(int argc, char** argv) {
  // Init cuda
  device_init(0);

  dim3 blocks(1);
  dim3 threads_per_block(32);
  test_kernel<<<blocks, threads_per_block>>>();
  CUDA_CHECK_ERROR(hipDeviceSynchronize());
}
